#include "hip/hip_runtime.h"
//----------//
// BLAS.cpp //
//-------------------------------------------------------//
// author: Jaegwang Lim @ Dexter Studios                 //
// last update: 2018.07.03                               //
//-------------------------------------------------------//

#include <Bora.h>

BORA_NAMESPACE_BEGIN namespace BLAS { // Basic Linear Algebra Subprogram

void mul( const FloatSparseMatrix& a, const FloatArray& b, FloatArray& c )
{
    const size_t rows = a.rows();
    const size_t cols = a.columns();

    if( rows != b.size() ) return;
    c.initialize( rows, b.memorySpace() );

    float* cp = c.pointer();

    auto kernel = [=] BORA_DEVICE ( const size_t r )
    {
        float* vals = a.valuesOnRow( r );
        size_t* inds = a.indicesOnRow( r );

        float acc = 0.f;

        for( size_t c=0; c<cols; ++ c )
        {
            size_t n = inds[c];
            if( n != NULL_MAX ) acc += vals[c] * b[n];
        }

        cp[r] = acc;
    };

    LaunchCudaDevice( kernel, 0, rows );
    SyncCuda();
}

void add( const FloatArray& a, const FloatArray& b, FloatArray& c )
{
    c.initialize( a.size(), a.memorySpace() );
    float* cp = c.pointer();

    auto kernel = [=] BORA_DEVICE ( const size_t n )
    {
        cp[n] = a[n] + b[n];
    };

    LaunchCudaDevice( kernel, 0, a.size() );
    SyncCuda();
}

void sub( const FloatArray& a, const FloatArray& b, FloatArray& c )
{
    c.initialize( a.size(), a.memorySpace() );
    float* cp = c.pointer();    
    
    auto kernel = [=] BORA_DEVICE ( const size_t n )
    {
        cp[n] = a[n] - b[n];
    };
    
    LaunchCudaDevice( kernel, 0, a.size() );
    SyncCuda();
}

void mul( const FloatArray& a, const FloatArray& b, FloatArray& c )
{
    c.initialize( a.size(), a.memorySpace() );
    float* cp = c.pointer();

    auto kernel = [=] BORA_DEVICE ( const size_t n )
    {   
        cp[n] = a[n] * b[n];
    };

    LaunchCudaDevice( kernel, 0, a.size() );
    SyncCuda();
}

void mul( const FloatArray& a, const float b, FloatArray& c )
{
    c.initialize( a.size(), a.memorySpace() );
    float* cp = c.pointer();
    
    auto kernel = [=] BORA_DEVICE ( const size_t n )
    {
        cp[n] = a[n] * b;
    };
    
    LaunchCudaDevice( kernel, 0, a.size() );
    SyncCuda();
}

float dot( const FloatArray& a, const FloatArray& b )
{
    thrust::device_ptr<float> d_a( a.pointer() );
    thrust::device_ptr<float> d_b( b.pointer() );

    return thrust::inner_product( d_a, d_a+a.size(), d_b, 0.f );
}

void equ( const FloatArray& a, FloatArray& c )
{
    c.initialize( a.size(), a.memorySpace() );
    for( size_t n=0; n<a.size(); ++n )
    {
        c[n] = a[n];
    }
}

float len( const FloatArray& a )
{
    float l(0.f);
    for( size_t n=0; n<a.size(); ++n )
    {
        l += a[n] * a[n];
    }
    return sqrt( l );
}

void addmul( const FloatArray& a, const float& alpha, const FloatArray& p, FloatArray& x )
{
    float* px = x.pointer();

    auto kernel = [=] BORA_DEVICE ( const size_t n )
    {
        px[n] = a[n] + (alpha * p[n]);
    };

    LaunchCudaDevice( kernel, 0, a.size() );
    SyncCuda();
}

void submul( const FloatArray& a, const float& alpha, const FloatArray& p, FloatArray& x )
{
    float* px = x.pointer();
    
    auto kernel = [=] BORA_DEVICE ( const size_t n )
    {
        px[n] = a[n] - (alpha * p[n]);
    };

    LaunchCudaDevice( kernel, 0, a.size() );
    SyncCuda();
}

void addmul( const FloatArray& a, const FloatSparseMatrix& A, const FloatArray& p, FloatArray& x )
{
    const size_t rows = A.rows();
    const size_t cols = A.columns();

    if( rows != p.size() ) return;
    x.initialize( rows, p.memorySpace() );

    float* px = x.pointer();

    auto kernel = [=] BORA_DEVICE ( const size_t r )
    {
        float* vals = A.valuesOnRow( r );
        size_t* inds = A.indicesOnRow( r );

        float acc = 0.f;

        for( size_t c=0; c<cols; ++ c )
        {
            size_t n = inds[c];
            if( n != NULL_MAX ) acc += vals[c] * p[n];
        }

        px[r] = a[r] + acc;
    };

    LaunchCudaDevice( kernel, 0, rows );
    SyncCuda();
}

void submul( const FloatArray& a, const FloatSparseMatrix& A, const FloatArray& p, FloatArray& x )
{
    const size_t rows = A.rows();
    const size_t cols = A.columns();

    if( rows != p.size() ) return;
    x.initialize( rows, p.memorySpace() );

    float* px = x.pointer();

    auto kernel = [=] BORA_DEVICE ( const size_t r )
    {
        float* vals = A.valuesOnRow( r );
        size_t* inds = A.indicesOnRow( r );

        float acc = 0.f;

        for( size_t c=0; c<cols; ++ c )
        {
            size_t n = inds[c];
            if( n != NULL_MAX ) acc += vals[c] * p[n];
        }

        px[r] = a[r] - acc;
    };

    LaunchCudaDevice( kernel, 0, rows );
    SyncCuda();
}

void buildPreconditioner( const SparseMatrix<float>& A, FloatArray& M )
{
    M.setValueAll( 0.f );
    float* pm = M.pointer();

    const float _micParam = 0.97f;    

    for( size_t n=0; n<A.rows(); ++n )    
    {
        const float* a_values = A.valuesOnRow( n );
        const size_t* a_indices = A.indicesOnRow( n );

        const size_t& a_i0jk = a_indices[1];
        const size_t& a_ij0k = a_indices[3];
        const size_t& a_ijk0 = a_indices[5];

        float e = a_values[0];
        float e2( 0.f );

        float alpha = e;

        if( a_i0jk != NULL_MAX )
        {
            const float* V = A.valuesOnRow(a_i0jk);
            
            e -= Pow2( V[2] * M[a_i0jk] );
            e2 += V[2] * ( V[4]+V[6] ) * Pow2( M[a_i0jk] );
        }        

        if( a_ij0k != NULL_MAX )
        {
            const float* V = A.valuesOnRow(a_ij0k);

            e -= Pow2( V[4] * M[a_ij0k] );
            e2 += V[4] * ( V[2]+V[6] ) * Pow2( M[a_ij0k] );
        }

        if( a_ijk0 != NULL_MAX )
        {
            const float* V = A.valuesOnRow(a_ijk0);

            e -= Pow2( V[6] * M[a_ijk0] );
            e2 += V[6] * ( V[2]+V[4] ) * Pow2( M[a_ijk0] );
        }

        e -= _micParam * e2;

        if( e < alpha * 0.25f ) e = alpha;

        pm[n] = 1.f / sqrt( e + EPSILON );
    };    
}

void applyPreconditioner( const SparseMatrix<float>& A, const FloatArray& M, const FloatArray& r, FloatArray& q, FloatArray& z )
{
    z.initialize( r.size(), r.memorySpace() );
    q.initialize( r.size(), r.memorySpace() );

    for( size_t n=0; n<r.size(); ++n )
    {        
        const float* a_values = A.valuesOnRow( n );
        const size_t* a_indices = A.indicesOnRow( n );

        size_t a_i0jk = a_indices[1];
        size_t a_ij0k = a_indices[3];
        size_t a_ijk0 = a_indices[5];

        float t = r[n];

        if( a_i0jk != NULL_MAX )
        {
            const float* V = A.valuesOnRow(a_i0jk);
            t -= V[2] * M[ a_i0jk ] * q[ a_i0jk ];
        }        

        if( a_ij0k != NULL_MAX )
        {
            const float* V = A.valuesOnRow(a_ij0k);
            t -= V[4] * M[ a_ij0k ] * q[ a_ij0k ];
        }

        if( a_ijk0 != NULL_MAX )
        {
            const float* V = A.valuesOnRow(a_ijk0);
            t -= V[6] * M[ a_ijk0 ] * q[ a_ijk0 ];
        }

        q[n] = t * M[n];
    }

    for( size_t l=0; l<r.size(); ++l )
    {
        size_t n = r.size()-1 - l;

        const float* a_values = A.valuesOnRow( n );
        const size_t* a_indices = A.indicesOnRow( n );

        size_t a_i1jk = a_indices[2];
        size_t a_ij1k = a_indices[4];
        size_t a_ijk1 = a_indices[6];

        float t = q[n];

        if( a_i1jk != NULL_MAX )
        { 
            t -= a_values[2] * M[n] * z[a_i1jk];
        }

        if( a_ij1k != NULL_MAX )
        {
            t -= a_values[4] * M[n] * z[a_ij1k];
        }

        if( a_ijk1 != NULL_MAX )
        {
            t -= a_values[6] * M[n] * z[a_ijk1];
        }
        
        z[n] = t * M[n];
    }    
}

} BORA_NAMESPACE_END

