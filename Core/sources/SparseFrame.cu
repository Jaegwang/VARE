#include "hip/hip_runtime.h"

#include <VARE.h>

VARE_NAMESPACE_BEGIN

SparseFrame* SparseFrame::create( const Grid& vGrid, MemorySpace mType )
{
    SparseFrame* newFrame=0;
    hipMallocManaged( &newFrame, sizeof(SparseFrame) );
    *newFrame = SparseFrame();

    newFrame->initialize( vGrid, mType );
    
    return newFrame;
}

void SparseFrame::remove( SparseFrame* frame )
{
    if( frame ) hipFree( frame );
}

void SparseFrame::initialize( const Grid& vGrid, MemorySpace mType )
{
    _memType = mType;

    const size_t block = _blockWidth;
    const size_t gx = vGrid.nx();
    const size_t gy = vGrid.ny();
    const size_t gz = vGrid.nz();

    _fx = (gx/block) + ((gx%block) ? 1 : 0);
    _fy = (gy/block) + ((gy%block) ? 1 : 0);
    _fz = (gz/block) + ((gz%block) ? 1 : 0);

    const size_t nx = _fx * block;
    const size_t ny = _fy * block;
    const size_t nz = _fz * block;

    Vec3f minP = vGrid.boundingBox().minPoint();
    Vec3f maxP = vGrid.boundingBox().maxPoint();
    Vec3f cenP = (minP + maxP)*0.5f;

    const float dx = (maxP.x-minP.x)/(float)gx;
    const float dy = (maxP.y-minP.y)/(float)gy;
    const float dz = (maxP.z-minP.z)/(float)gz;
    const float h = Min( Min(dx,dy),dz );

    const float lx = (float)nx * h;
    const float ly = (float)ny * h;
    const float lz = (float)nz * h;

    minP = cenP - Vec3f( lx*0.5f, ly*0.5f, lz*0.5f );
    maxP = minP + Vec3f( lx, ly, lz );

    size_t num = _fx*_fy*_fz;

	_fxfy = _fx*_fy;
	_br = _blockWidth;
	_brbr = _br*_br;
	_brbrbr = _br*_br*_br;

    _pointers.initialize( num, kUnified );
    _pointers.setValueAll( INVALID_MAX );

    _markers.initialize( num, kUnified );
    _markers.setValueAll( 0 );

    _coords.initialize( 0, kUnified );
    _coords.reserve( num/block+1 );

    _grid.initialize( nx, ny, nz, minP, maxP );
}

void SparseFrame::build()
{
    for( auto it=_map.begin(); it!=_map.end(); ++it )
    {
        it->second->build();
    }
}

void SparseFrame::buildFromPoints( const PointArray& points, const bool enableTank, const float height )
{
    const int fx = _fx;
    const int fy = _fy;
    const int fz = _fz;
    const int block_res = _blockWidth;
    const int block_size = _blockWidth*_blockWidth*_blockWidth;

    const float hh = (float)block_res*0.5f;

    _markers.zeroize();
    int* pMark = _markers.pointer();

    auto kernel = VARE_DEVICE_KERNEL
    {
        const Vec3f& p = points[ix];
        
        int ci = Max( (int)(p.x)/block_res, 0 );
        int cj = Max( (int)(p.y)/block_res, 0 );
        int ck = Max( (int)(p.z)/block_res, 0 );

        for( int k=ck-1; k<=Min(ck+1,fz-1); ++k )
        for( int j=cj-1; j<=Min(cj+1,fy-1); ++j )
        for( int i=ci-1; i<=Min(ci+1,fx-1); ++i )
        {
            size_t t = k*fx*fy + j*fx + i;

            int& m = pMark[t];
            atomicAdd( &m, 1 );
        }
    };

    LaunchDeviceKernel( kernel, 0, points.size() );
    SyncKernels();

    if( enableTank == true )
    {
        const int tall = Clamp( (int)(height)/block_res, 0, (int)_fy );
        for( int k=0; k<_fz  ; ++k )
        for( int j=0; j< tall; ++j )
        for( int i=0; i<_fx  ; ++i )
        {
            size_t t = k*fx*fy + j*fx + i;
            pMark[t] = 1;
        }
    }

    _coords.clear();

    size_t count = 0;
    for( size_t n=0; n<_markers.size(); ++n )
    {
        if( _markers[n] > 0 )
        {
            _pointers[n] = count;

            Idx3 coord;
            coord.i = (n) % _fx;
            coord.j = (n/_fx) % _fy;
            coord.k = (n/(_fx*_fy)) % _fz;
            _coords.append( coord );

            count += block_size;
        }
        else
        {
            _pointers[n] = INVALID_MAX;
        }
    }

    build();
}

VARE_NAMESPACE_END

